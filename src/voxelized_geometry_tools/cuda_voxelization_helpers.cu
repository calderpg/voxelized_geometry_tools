#include "hip/hip_runtime.h"
#include <voxelized_geometry_tools/cuda_voxelization_helpers.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstring>
#include <map>
#include <iostream>
#include <string>
#include <vector>

namespace voxelized_geometry_tools
{
namespace pointcloud_voxelization
{
namespace cuda_helpers
{
void CudaCheckErrors(const std::string& msg)
{
  const hipError_t last_error = hipGetLastError();
  if (last_error != hipSuccess)
  {
    const std::string error_string(hipGetErrorString(last_error));
    throw std::runtime_error("[" + msg + "] Cuda error [" + error_string + "]");
  }
}

__global__
void RaycastPoint(
    const float* const device_points_ptr, const int32_t num_points,
    const float max_range,
    const float* const device_grid_pointcloud_transform_ptr,
    const float inverse_step_size, const float inverse_cell_size,
    const int32_t num_x_cells, const int32_t num_y_cells,
    const int32_t num_z_cells, const int32_t stride1, const int32_t stride2,
    int32_t* const device_tracking_grid_ptr)
{
  const int32_t point_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (point_index < num_points)
  {
    // Point in pointcloud frame
    const float px = device_points_ptr[(point_index * 3) + 0];
    const float py = device_points_ptr[(point_index * 3) + 1];
    const float pz = device_points_ptr[(point_index * 3) + 2];
    // Skip invalid points marked with NaN or infinity
    if (isfinite(px) && isfinite(py) && isfinite(pz))
    {
      // Pointcloud origin in grid frame
      const float ox = device_grid_pointcloud_transform_ptr[12];
      const float oy = device_grid_pointcloud_transform_ptr[13];
      const float oz = device_grid_pointcloud_transform_ptr[14];
      // Point in grid frame
      const float gx = device_grid_pointcloud_transform_ptr[0] * px
                       + device_grid_pointcloud_transform_ptr[4] * py
                       + device_grid_pointcloud_transform_ptr[8] * pz
                       + device_grid_pointcloud_transform_ptr[12];
      const float gy = device_grid_pointcloud_transform_ptr[1] * px
                       + device_grid_pointcloud_transform_ptr[5] * py
                       + device_grid_pointcloud_transform_ptr[9] * pz
                       + device_grid_pointcloud_transform_ptr[13];
      const float gz = device_grid_pointcloud_transform_ptr[2] * px
                       + device_grid_pointcloud_transform_ptr[6] * py
                       + device_grid_pointcloud_transform_ptr[10] * pz
                       + device_grid_pointcloud_transform_ptr[14];
      const float rx = gx - ox;
      const float ry = gy - oy;
      const float rz = gz - oz;
      const float current_ray_length = sqrtf((rx * rx) + (ry * ry) + (rz * rz));
      const float num_steps = floor(current_ray_length * inverse_step_size);
      int32_t previous_x_cell = -1;
      int32_t previous_y_cell = -1;
      int32_t previous_z_cell = -1;
      bool ray_crossed_grid = false;
      for (float step = 0.0; step < num_steps; step += 1.0)
      {
        const float elapsed_ratio = step / num_steps;
        if ((elapsed_ratio * current_ray_length) > max_range)
        {
          // We've gone beyond max range of the sensor
          break;
        }
        const float qx = (rx * elapsed_ratio) + ox;
        const float qy = (ry * elapsed_ratio) + oy;
        const float qz = (rz * elapsed_ratio) + oz;
        const int32_t x_cell =
            static_cast<int32_t>(std::floor(qx * inverse_cell_size));
        const int32_t y_cell =
            static_cast<int32_t>(std::floor(qy * inverse_cell_size));
        const int32_t z_cell =
            static_cast<int32_t>(std::floor(qz * inverse_cell_size));
        if (x_cell != previous_x_cell || y_cell != previous_y_cell
            || z_cell != previous_z_cell)
        {
          if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
              && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
          {
            ray_crossed_grid = true;
            const int32_t cell_index =
                (x_cell * stride1) + (y_cell * stride2) + z_cell;
            // Increase free count
            atomicAdd(&(device_tracking_grid_ptr[cell_index * 2]), 1);
          }
          else if (ray_crossed_grid)
          {
            // We've left the grid and there's no reason to keep going.
            break;
          }
        }
        previous_x_cell = x_cell;
        previous_y_cell = y_cell;
        previous_z_cell = z_cell;
      }
      // Set the point itself as filled, if it is in range
      if (current_ray_length <= max_range)
      {
        const int32_t x_cell =
            static_cast<int32_t>(std::floor(gx * inverse_cell_size));
        const int32_t y_cell =
            static_cast<int32_t>(std::floor(gy * inverse_cell_size));
        const int32_t z_cell =
            static_cast<int32_t>(std::floor(gz * inverse_cell_size));
        if (x_cell >= 0 && x_cell < num_x_cells && y_cell >= 0
            && y_cell < num_y_cells && z_cell >= 0 && z_cell < num_z_cells)
        {
          const int32_t cell_index =
              (x_cell * stride1) + (y_cell * stride2) + z_cell;
          // Increase filled count
          atomicAdd(&(device_tracking_grid_ptr[(cell_index * 2) + 1]), 1);
        }
      }
    }
  }
}

__global__
void FilterGrids(
    const int64_t num_cells, const int32_t num_grids,
    const int32_t* const device_tracking_grids_ptr,
    float* const device_filter_grid_ptr, const float percent_seen_free,
    const int32_t outlier_points_threshold, const int32_t num_cameras_seen_free)
{
  const int32_t voxel_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (voxel_index < num_cells)
  {
    const float current_occupancy = device_filter_grid_ptr[voxel_index * 2];
    // Filled cells stay filled, we don't work with them.
    // We only change cells that are unknown or empty.
    if (current_occupancy <= 0.5)
    {
      int32_t cameras_seen_filled = 0;
      int32_t cameras_seen_free = 0;
      for (int32_t idx = 0; idx < num_grids; idx++)
      {
        const int32_t* const device_tracking_grid_ptr =
            device_tracking_grids_ptr + (idx * num_cells * 2);
        const int32_t free_count = device_tracking_grid_ptr[voxel_index * 2];
        const int32_t filled_count =
            device_tracking_grid_ptr[(voxel_index * 2) + 1];
        const int32_t filtered_filled_count =
            (filled_count >= outlier_points_threshold) ? filled_count : 0;
        if (free_count > 0 && filtered_filled_count > 0)
        {
          const float current_percent_seen_free =
              static_cast<float>(free_count)
              / static_cast<float>(free_count + filtered_filled_count);
          if (current_percent_seen_free >= percent_seen_free)
          {
            cameras_seen_free += 1;
          }
          else
          {
            cameras_seen_filled += 1;
          }
        }
        else if (free_count > 0)
        {
          cameras_seen_free += 1;
        }
        else if (filtered_filled_count > 0)
        {
          cameras_seen_filled += 1;
        }
      }
      if (cameras_seen_filled > 0)
      {
        // If any camera saw something here, it is filled.
        device_filter_grid_ptr[voxel_index * 2] = 1.0;
      }
      else if (cameras_seen_free >= num_cameras_seen_free)
      {
        // Did enough cameras see this empty?
        device_filter_grid_ptr[voxel_index * 2] = 0.0;
      }
      else
      {
        // Otherwise, it is unknown.
        device_filter_grid_ptr[voxel_index * 2] = 0.5;
      }
    }
  }
}

class CudaTrackingGridsHandle : public TrackingGridsHandle
{
public:
  CudaTrackingGridsHandle(
      int32_t* const tracking_grids_buffer,
      const std::vector<int64_t>& tracking_grid_starting_offsets,
      const int64_t num_cells_per_grid)
      : TrackingGridsHandle(tracking_grid_starting_offsets, num_cells_per_grid),
        tracking_grids_buffer_(tracking_grids_buffer)
  {
    if (tracking_grids_buffer_ == nullptr)
    {
      throw std::invalid_argument(
          "Cannot create CudaTrackingGridsHandle with null buffer");
    }
  }

  ~CudaTrackingGridsHandle() override
  {
    hipFree(tracking_grids_buffer_);
    CudaCheckErrors("Failed to free device tracking grids buffer");
    tracking_grids_buffer_ = nullptr;
  }

  int32_t* GetBuffer() const { return tracking_grids_buffer_; }

private:
  int32_t* tracking_grids_buffer_ = nullptr;
};

class CudaFilterGridHandle : public FilterGridHandle
{
public:
  CudaFilterGridHandle(
      float* const filter_grid_buffer, const int64_t num_cells)
      : FilterGridHandle(num_cells), filter_grid_buffer_(filter_grid_buffer)
  {
    if (filter_grid_buffer_ == nullptr)
    {
      throw std::invalid_argument(
          "Cannot create CudaFilterGridHandle with null buffer");
    }
  }

  ~CudaFilterGridHandle() override
  {
    hipFree(filter_grid_buffer_);
    CudaCheckErrors("Failed to free device filter grid buffer");
    filter_grid_buffer_ = nullptr;
  }

  float* GetBuffer() const { return filter_grid_buffer_; }

private:
  float* filter_grid_buffer_ = nullptr;
};

class CudaVoxelizationHelperInterface : public DeviceVoxelizationHelperInterface
{
public:
  explicit CudaVoxelizationHelperInterface(
      const std::map<std::string, int32_t>& options)
  {
    const int32_t cuda_device =
        RetrieveOptionOrDefault(options, "CUDA_DEVICE", 0);
    try
    {
      int32_t device_count = 0;
      hipGetDeviceCount(&device_count);
      CudaCheckErrors("Failed to get device count");
      if (cuda_device >= 0 && cuda_device < device_count)
      {
        cuda_device_num_ = cuda_device;
        SetCudaDevice();

        hipDeviceProp_t device_properties;
        std::memset(&device_properties, 0, sizeof(device_properties));
        hipGetDeviceProperties(&device_properties, cuda_device);
        CudaCheckErrors("Failed to get device properties");
        const std::string device_name(device_properties.name);

        std::cout << "Using CUDA device [" << cuda_device << "] - Name: ["
                  << device_name << "]" << std::endl;
      }
      else
      {
        std::cerr << "CUDA_DEVICE = " << cuda_device << " out of range for "
                  << device_count << " devices" << std::endl;
        cuda_device_num_ = -1;
      }
    }
    catch (const std::runtime_error& ex)
    {
      std::cerr << "Failed to load CUDA runtime and set device: "
                << ex.what() << std::endl;
      cuda_device_num_ = -1;
    }
  }

  bool IsAvailable() const override { return (cuda_device_num_ >= 0); }

  std::unique_ptr<TrackingGridsHandle> PrepareTrackingGrids(
      const int64_t num_cells, const int32_t num_grids) override
  {
    const size_t tracking_grids_size =
        sizeof(int32_t) * 2 * num_cells * num_grids;
    int32_t* tracking_grids_buffer = nullptr;
    hipMalloc(&tracking_grids_buffer, tracking_grids_size);
    CudaCheckErrors("Failed to allocate device tracking grids");
    hipMemset(tracking_grids_buffer, 0, tracking_grids_size);
    CudaCheckErrors("Failed to zero device tracking grids");

    std::vector<int64_t> tracking_grid_offsets(num_grids, 0);
    for (int32_t num_grid = 0; num_grid < num_grids; num_grid++)
    {
      tracking_grid_offsets.at(num_grid) = num_grid * num_cells * 2;
    }

    return std::unique_ptr<TrackingGridsHandle>(
        new CudaTrackingGridsHandle(
            tracking_grids_buffer, tracking_grid_offsets, num_cells));
  }

  void RaycastPoints(
      const std::vector<float>& raw_points, const float max_range,
      const float* const grid_pointcloud_transform,
      const float inverse_step_size, const float inverse_cell_size,
      const int32_t num_x_cells, const int32_t num_y_cells,
      const int32_t num_z_cells, TrackingGridsHandle& tracking_grids,
      const size_t tracking_grid_index) override
  {
    CudaTrackingGridsHandle& real_tracking_grids =
        dynamic_cast<CudaTrackingGridsHandle&>(tracking_grids);

    SetCudaDevice();
    const int32_t num_points = raw_points.size() / 3;
    // Copy the points
    const size_t points_size = sizeof(float) * raw_points.size();
    float* device_points_ptr = nullptr;
    hipMalloc(&device_points_ptr, points_size);
    CudaCheckErrors("Failed to allocate device points");
    hipMemcpy(device_points_ptr, raw_points.data(), points_size,
               hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the points to the device");

    // Copy grid pointcloud transform
    const size_t transform_size = sizeof(float) * 16;
    float* device_grid_pointcloud_transform_ptr = nullptr;
    hipMalloc(&device_grid_pointcloud_transform_ptr, transform_size);
    CudaCheckErrors("Failed to allocate device grid pointcloud transform");
    hipMemcpy(
        device_grid_pointcloud_transform_ptr, grid_pointcloud_transform,
        transform_size, hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the grid pointcloud transform");

    // Prepare for raycasting
    const int32_t stride1 = num_y_cells * num_z_cells;
    const int32_t stride2 = num_z_cells;
    // Call the CUDA kernel
    const int32_t num_threads = 256;
    const int32_t num_blocks = (num_points + (num_threads - 1)) / num_threads;
    const size_t starting_index =
        real_tracking_grids.GetTrackingGridStartingOffset(tracking_grid_index);
    int32_t* const device_tracking_grid_ptr =
        real_tracking_grids.GetBuffer() + starting_index;
    RaycastPoint<<<num_blocks, num_threads>>>(
        device_points_ptr, num_points, max_range,
        device_grid_pointcloud_transform_ptr, inverse_step_size,
        inverse_cell_size, num_x_cells, num_y_cells, num_z_cells, stride1,
        stride2, device_tracking_grid_ptr);

    // Free the device memory
    hipFree(device_points_ptr);
    CudaCheckErrors("Failed to free device points");
    hipFree(device_grid_pointcloud_transform_ptr);
    CudaCheckErrors("Failed to free device grid pointcloud transform");
  }

  std::unique_ptr<FilterGridHandle> PrepareFilterGrid(
       const int64_t num_cells, const void* host_data_ptr) override
  {
    const size_t filter_grid_size = sizeof(float) * num_cells * 2;
    float* filter_grid_buffer = nullptr;
    hipMalloc(&filter_grid_buffer, filter_grid_size);
    CudaCheckErrors("Failed to allocate device filter grid");
    hipMemcpy(filter_grid_buffer, host_data_ptr, filter_grid_size,
               hipMemcpyHostToDevice);
    CudaCheckErrors("Failed to memcpy the static environment to the device");

    return std::unique_ptr<FilterGridHandle>(new CudaFilterGridHandle(
        filter_grid_buffer, num_cells));
  }

  void FilterTrackingGrids(
      const TrackingGridsHandle& tracking_grids, const float percent_seen_free,
      const int32_t outlier_points_threshold,
      const int32_t num_cameras_seen_free,
      FilterGridHandle& filter_grid) override
  {
    const CudaTrackingGridsHandle& real_tracking_grids =
        dynamic_cast<const CudaTrackingGridsHandle&>(tracking_grids);
    CudaFilterGridHandle& real_filter_grid =
        dynamic_cast<CudaFilterGridHandle&>(filter_grid);

    // Call the CUDA kernel
    const int32_t num_threads = 256;
    const int32_t num_blocks =
        (real_tracking_grids.NumCellsPerGrid() + (num_threads - 1))
        / num_threads;
    FilterGrids<<<num_blocks, num_threads>>>(
        real_tracking_grids.NumCellsPerGrid(),
        static_cast<int32_t>(real_tracking_grids.GetNumTrackingGrids()),
        real_tracking_grids.GetBuffer(), real_filter_grid.GetBuffer(),
        percent_seen_free, outlier_points_threshold, num_cameras_seen_free);
  }

  void RetrieveTrackingGrid(
      const TrackingGridsHandle& tracking_grids,
      const size_t tracking_grid_index, void* host_data_ptr) override
  {
    const CudaTrackingGridsHandle& real_tracking_grids =
        dynamic_cast<const CudaTrackingGridsHandle&>(tracking_grids);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    const size_t item_size = sizeof(int32_t) * 2;
    const size_t tracking_grid_size =
        real_tracking_grids.NumCellsPerGrid() * item_size;
    const size_t starting_index =
        real_tracking_grids.GetTrackingGridStartingOffset(tracking_grid_index);
    hipMemcpy(host_data_ptr,
               real_tracking_grids.GetBuffer() + starting_index,
               tracking_grid_size, hipMemcpyDeviceToHost);
    CudaCheckErrors("Failed to memcpy the tracking grid back to the host");
  }

  void RetrieveFilteredGrid(
      const FilterGridHandle& filter_grid, void* host_data_ptr) override
  {
    const CudaFilterGridHandle& real_filter_grid =
        dynamic_cast<const CudaFilterGridHandle&>(filter_grid);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    const size_t item_size = sizeof(float) * 2;
    const size_t buffer_size = real_filter_grid.NumCells() * item_size;
    hipMemcpy(host_data_ptr, real_filter_grid.GetBuffer(), buffer_size,
               hipMemcpyDeviceToHost);
    CudaCheckErrors("Failed to memcpy the filter grid back to the host");
  }

  void SetCudaDevice()
  {
    hipSetDevice(cuda_device_num_);
    CudaCheckErrors("Failed to set device");
  }

private:
  int32_t cuda_device_num_ = -1;
};

std::vector<AvailableDevice> GetAvailableDevices()
{
  std::vector<AvailableDevice> available_devices;

  try
  {
    int32_t device_count = 0;
    hipGetDeviceCount(&device_count);
    CudaCheckErrors("Failed to get device count");

    for (int32_t device_idx = 0; device_idx < device_count; device_idx++)
    {
      hipDeviceProp_t device_properties;
      std::memset(&device_properties, 0, sizeof(device_properties));
      hipGetDeviceProperties(&device_properties, device_idx);
      CudaCheckErrors("Failed to get device properties");
      const std::string device_name(device_properties.name);
      const std::string full_name = "CUDA - Device: [" + device_name + "]";

      std::map<std::string, int32_t> device_options;
      device_options["CUDA_DEVICE"] = device_idx;

      available_devices.push_back(AvailableDevice(full_name, device_options));
    }
  }
  catch (const std::runtime_error& ex)
  {
    std::cerr << ex.what() << std::endl;
  }

  return available_devices;
}

std::unique_ptr<DeviceVoxelizationHelperInterface>
MakeCudaVoxelizationHelper(const std::map<std::string, int32_t>& options)
{
  return std::unique_ptr<DeviceVoxelizationHelperInterface>(
      new CudaVoxelizationHelperInterface(options));
}
}  // namespace cuda_helpers
}  // namespace pointcloud_voxelization
}  // namespace voxelized_geometry_tools
